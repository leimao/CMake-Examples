#include "hip/hip_runtime.h"
#include "kernels.h"
#include <iostream>

__global__
void addKernel(int n, float * x, float * y)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
    {
        y[i] = x[i] + y[i];
    }
}

void runAddKernel(int n)
{
    float * x, * y;

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&x, n * sizeof(float));
    hipMallocManaged(&y, n * sizeof(float));
    
    // initialize x and y arrays on the host
    for (int i = 0; i < n; i++) 
    {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Run the kernel
    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;
    // add<<<1, blockSize>>>(N, x, y);
    addKernel<<<gridSize, blockSize>>>(n, x, y);
    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < n; i++)
        maxError = fmax(maxError, fabs(y[i]-3.0f));
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);
}